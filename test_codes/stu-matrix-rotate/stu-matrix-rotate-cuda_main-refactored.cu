#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void rotate_matrix_parallel (float *matrix, const int n) {
  int layer = blockIdx.x * blockDim.x + threadIdx.x;
  if (layer < n/2) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;

      float top = matrix[first*n+i]; // save top
      // left -> top
      matrix[first*n+i] = matrix[(last-offset)*n+first];

      // bottom -> left
      matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

      // right -> bottom
      matrix[last*n+(last-offset)] = matrix[i*n+last];

      // top -> right
      matrix[i*n+last] = top; // right <- saved top
    }
  }
}


int main(int argc, char** argv) {
  if (argc != 3) {
    printf("Usage: %s <matrix size> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

    float *parallel_res = (float*) aligned_alloc(1024, n*n*sizeof(float));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      parallel_res[i*n+j] = i*n+j;

  float *d_parallel_res;
  hipMalloc((void**)&d_parallel_res, n*n*sizeof(float));
  hipMemcpy(d_parallel_res, parallel_res, n*n*sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  dim3 threadsPerBlock(64, 64);
  dim3 numBlocks((n + threadsPerBlock.x -1) / threadsPerBlock.x, (n+threadsPerBlock.y -1) / threadsPerBlock.y);
  
  for (int i = 0; i < repeat; i++) {
    // rotate_matrix_parallel<<<(n + threadsPerBlock.x - 1)/ threadsPerBlock.x, (n + threadsPerBlock.y - 1) / threadsPerBlock.y>>>(d_parallel_res, n);
    // rotate_matrix_parallel<<<(n +255)/256, 256>>>(d_parallel_res, n);
    // rotate_matrix_parallel<<<numBlocks, threadsPerBlock>>>(d_parallel_res, n);
    rotate_matrix_parallel<<<(n +255)/256, 64>>>(d_parallel_res, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(parallel_res, d_parallel_res, n*n*sizeof(float), hipMemcpyDeviceToHost);

  free(parallel_res);
  hipFree(d_parallel_res);
  return 0;
}
